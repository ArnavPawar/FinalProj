#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define num_trees 256
#define connections 5
#define spread_chance 0.3
#define extinguish_chance 0.1
#define startBurningChance 0.05

struct Cell {
    int status;  // 0 = empty, 1 = tree, 2 = burning
    int neighbors[connections];
};

__global__ void setupRNG(hiprandState *state, unsigned long seed) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void initForest(Cell *forest, hiprandState *states) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < num_trees) {
        hiprandState localState = states[id];
        forest[id].status = (hiprand_uniform(&localState) < startBurningChance) ? 2 : 1;
        for (int i = 0; i < connections; i++) {
            forest[id].neighbors[i] = hiprand(&localState) % num_trees;
        }
        states[id] = localState;
    }
}

__global__ void spreadFire(Cell *forest, hiprandState *states, int *changes) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < num_trees && forest[id].status == 2) {
        hiprandState localState = states[id];
        for (int i = 0; i < connections; i++) {
            int nid = forest[id].neighbors[i];
            if (forest[nid].status == 1 && hiprand_uniform(&localState) < spread_chance) {
                atomicExch(&forest[nid].status, 2);
                atomicAdd(&changes[0], 1);
            }
        }
        if (hiprand_uniform(&localState) < extinguish_chance) {
            forest[id].status = 0;
            atomicAdd(&changes[1], 1);
        }
        states[id] = localState;
    }
}

int main() {
    Cell *d_forest;
    hiprandState *d_states;
    int *d_changes;
    int changes[2] = {0, 0};

    hipMalloc(&d_forest, num_trees * sizeof(Cell));
    hipMalloc(&d_states, num_trees * sizeof(hiprandState));
    hipMalloc(&d_changes, 2 * sizeof(int));
    hipMemcpy(d_changes, changes, 2 * sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks((num_trees + 255) / 256);
    dim3 threads(256);

    setupRNG<<<blocks, threads>>>(d_states, time(NULL));
    initForest<<<blocks, threads>>>(d_forest, d_states);
    hipDeviceSynchronize();

    for (int day = 0; day < 100; day++) {
        hipMemcpy(d_changes, changes, 2 * sizeof(int), hipMemcpyHostToDevice);
        spreadFire<<<blocks, threads>>>(d_forest, d_states, d_changes);
        hipDeviceSynchronize();
        hipMemcpy(changes, d_changes, 2 * sizeof(int), hipMemcpyDeviceToHost);
        if (changes[0] == 0 && changes[1] == 0) break; // No new fires and no new extinguishments
        printf("Day %d: New Fires: %d, Extinguished: %d\n", day, changes[0], changes[1]);
        changes[0] = 0;
        changes[1] = 0;
    }

    hipFree(d_forest);
    hipFree(d_states);
    hipFree(d_changes);
    return 0;
}
